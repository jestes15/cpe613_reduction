#include "reduction_algo.cuh"

#include <algorithm>
#include <chrono>
#include <iomanip>
#include <iostream>

template <typename datatype> void print_data(const char *kernel_name, datatype result, float time)
{
    if constexpr (std::is_same<float, datatype>::value)
        printf("%s: %f, %f ms\n", kernel_name, result, time);
    else if constexpr (std::is_same<double, datatype>::value)
        printf("%s: %f, %f ms\n", kernel_name, result, time);
}

template <typename datatype> int get_max_test_shift()
{
    if constexpr (std::is_same<float, datatype>::value)
        return 30;
    else if constexpr (std::is_same<double, datatype>::value)
        return 30;
}

template <typename datatype> void run_tests()
{
    if constexpr (std::is_same<float, datatype>::value)
        printf("SINGLE PRECISION TESTING\n");
    else if constexpr (std::is_same<double, datatype>::value)
        printf("DOUBLE PRECISION TESTING\n");

    float milliseconds = 0;

    int block_kernel2, grid_kernel2;
    int block_kernel3, grid_kernel3;
    int block_kernel4, grid_kernel4;
    int block_kernel5, grid_kernel5;
    int block_kernel6, grid_kernel6;

    std::vector<std::pair<uint64_t, float>> time_kernel1;
    std::vector<std::pair<uint64_t, float>> time_kernel2;
    std::vector<std::pair<uint64_t, float>> time_kernel3;
    std::vector<std::pair<uint64_t, float>> time_kernel4;
    std::vector<std::pair<uint64_t, float>> time_kernel5;
    std::vector<std::pair<uint64_t, float>> time_kernel6;
    std::vector<std::pair<uint64_t, float>> not_threaded;
    std::vector<std::pair<uint64_t, float>> threaded;
    std::vector<std::pair<uint64_t, float>> cub;
    std::vector<std::pair<uint64_t, float>> thrust;

    for (int i = 0; i < get_max_test_shift<datatype>(); ++i)
    {
        uint64_t size = (uint64_t)2 << (uint64_t)i;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        datatype *input_array = (datatype *)malloc(sizeof(datatype) * size);
        datatype zero = 0;
        std::fill(input_array, input_array + size, 1.5);

        std::cout << "-------------------------------------------------------------------\n";
        printf("Input Size: %lu (%f GB)\n", size, (size * sizeof(datatype)) / (float)1e9);

        const auto start_no_threading = std::chrono::steady_clock::now();
        datatype output_no_threading = host_reduction(input_array, size);
        const auto end_no_threading = std::chrono::steady_clock::now();
        const std::chrono::duration<double> diff_no_threading = end_no_threading - start_no_threading;
        print_data("NO THREAD", output_no_threading,
                   std::chrono::duration_cast<std::chrono::nanoseconds>(diff_no_threading).count() /
                       static_cast<float>(1e6));

        const auto start_threading = std::chrono::steady_clock::now();
        datatype output_threading = host_openmp_reduction(input_array, size);
        const auto end_threading = std::chrono::steady_clock::now();
        const std::chrono::duration<double> diff_threading = end_threading - start_threading;
        print_data("THREAD", output_threading,
                   std::chrono::duration_cast<std::chrono::nanoseconds>(diff_threading).count() /
                       static_cast<float>(1e6));

        datatype *d_input, *d_output_kernel1, *d_output_kernel2, *d_output_kernel3, *d_output_kernel4,
            *d_output_kernel5, *d_output_kernel6, *d_cub_output, *d_thrust_output;
        datatype output_kernel1, output_kernel2, output_kernel3, output_kernel4, output_kernel5, output_kernel6,
            cub_output, thrust_output;

        hipMalloc((void **)&d_input, sizeof(datatype) * size);
        hipMalloc((void **)&d_output_kernel1, sizeof(datatype));
        hipMalloc((void **)&d_output_kernel2, sizeof(datatype));
        hipMalloc((void **)&d_output_kernel3, sizeof(datatype));
        hipMalloc((void **)&d_output_kernel4, sizeof(datatype));
        hipMalloc((void **)&d_output_kernel5, sizeof(datatype));
        hipMalloc((void **)&d_output_kernel6, sizeof(datatype));
        hipMalloc((void **)&d_cub_output, sizeof(datatype));
        hipMalloc((void **)&d_thrust_output, sizeof(datatype));

        hipMemcpy(d_input, input_array, sizeof(datatype) * size, hipMemcpyHostToDevice);
        hipMemcpy(d_output_kernel1, &zero, sizeof(datatype), hipMemcpyHostToDevice);
        hipMemcpy(d_output_kernel2, &zero, sizeof(datatype), hipMemcpyHostToDevice);
        hipMemcpy(d_output_kernel3, &zero, sizeof(datatype), hipMemcpyHostToDevice);
        hipMemcpy(d_output_kernel4, &zero, sizeof(datatype), hipMemcpyHostToDevice);
        hipMemcpy(d_output_kernel5, &zero, sizeof(datatype), hipMemcpyHostToDevice);
        hipMemcpy(d_output_kernel6, &zero, sizeof(datatype), hipMemcpyHostToDevice);
        hipMemcpy(d_cub_output, &zero, sizeof(datatype), hipMemcpyHostToDevice);
        hipMemcpy(d_thrust_output, &zero, sizeof(datatype), hipMemcpyHostToDevice);

        hipEventRecord(start);
        reduce_kernel1<<<1, 1>>>(d_output_kernel1, d_input, size);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipMemcpy(&output_kernel1, d_output_kernel1, sizeof(datatype), hipMemcpyDeviceToHost);
        hipEventElapsedTime(&milliseconds, start, stop);
        print_data("KERNEL 1", output_kernel1, milliseconds);

        block_kernel2 = 1024;
        grid_kernel2 = (size + block_kernel2 - 1) / block_kernel2;

        hipEventRecord(start);
        reduce_kernel2<<<grid_kernel2, block_kernel2>>>(d_output_kernel2, d_input, size);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipMemcpy(&output_kernel2, d_output_kernel2, sizeof(datatype), hipMemcpyDeviceToHost);
        hipMemcpy(d_input, input_array, sizeof(datatype) * size, hipMemcpyHostToDevice);
        hipEventElapsedTime(&milliseconds, start, stop);
        print_data("KERNEL 2", output_kernel2, milliseconds);

        block_kernel3 = 1024;
        grid_kernel3 = (size + block_kernel3 - 1) / block_kernel3;

        hipEventRecord(start);
        reduce_kernel3<<<grid_kernel3, block_kernel3>>>(d_output_kernel3, d_input, size);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipMemcpy(&output_kernel3, d_output_kernel3, sizeof(datatype), hipMemcpyDeviceToHost);
        hipMemcpy(d_input, input_array, sizeof(datatype) * size, hipMemcpyHostToDevice);
        hipEventElapsedTime(&milliseconds, start, stop);
        print_data("KERNEL 3", output_kernel3, milliseconds);

        block_kernel4 = 1024;
        grid_kernel4 = (size + block_kernel4 - 1) / block_kernel4;

        hipEventRecord(start);
        reduce_kernel4<<<grid_kernel4, block_kernel4>>>(d_output_kernel4, d_input, size);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipMemcpy(&output_kernel4, d_output_kernel4, sizeof(datatype), hipMemcpyDeviceToHost);
        hipMemcpy(d_input, input_array, sizeof(datatype) * size, hipMemcpyHostToDevice);
        hipEventElapsedTime(&milliseconds, start, stop);
        print_data("KERNEL 4", output_kernel4, milliseconds);

        block_kernel5 = 1024;
        grid_kernel5 = (size + block_kernel5 - 1) / block_kernel5;

        hipEventRecord(start);
        reduce_kernel5<<<grid_kernel5, block_kernel5>>>(d_output_kernel5, d_input, size);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipMemcpy(&output_kernel5, d_output_kernel5, sizeof(datatype), hipMemcpyDeviceToHost);
        hipMemcpy(d_input, input_array, sizeof(datatype) * size, hipMemcpyHostToDevice);
        hipEventElapsedTime(&milliseconds, start, stop);
        print_data("KERNEL 5", output_kernel5, milliseconds);

        block_kernel6 = 1024;
        grid_kernel6 = (size + block_kernel6 - 1) / block_kernel6;

        for (int course_factor = 1; course_factor < 17; ++course_factor)
        {
            hipEventRecord(start);
            reduce_kernel6<datatype><<<grid_kernel6, block_kernel6>>>(d_output_kernel6, d_input, size, course_factor);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipMemcpy(&output_kernel6, d_output_kernel6, sizeof(datatype), hipMemcpyDeviceToHost);
            hipMemcpy(d_input, input_array, sizeof(datatype) * size, hipMemcpyHostToDevice);
            hipEventElapsedTime(&milliseconds, start, stop);
            hipMemcpy(d_output_kernel6, &zero, sizeof(datatype), hipMemcpyHostToDevice);
            print_data("KERNEL 6", output_kernel6, milliseconds);
        }

        auto cub_time = run_cub_reduce(d_cub_output, d_input, size);
        hipMemcpy(&cub_output, d_cub_output, sizeof(datatype), hipMemcpyDeviceToHost);
        hipMemcpy(d_input, input_array, sizeof(datatype) * size, hipMemcpyHostToDevice);
        print_data("CUB::REDUCE", cub_output, cub_time);

        auto thrust_time = run_thrust_reduce(d_thrust_output, d_input, size);
        hipMemcpy(&thrust_output, d_thrust_output, sizeof(datatype), hipMemcpyDeviceToHost);
        print_data("THRUST::REDUCE", thrust_output, thrust_time);

        free(input_array);

        hipFree(d_input);
        hipFree(d_output_kernel1);
        hipFree(d_output_kernel2);
        hipFree(d_output_kernel3);
        hipFree(d_output_kernel4);
        hipFree(d_output_kernel5);
        hipFree(d_output_kernel6);
        hipFree(d_cub_output);

        std::cout << "-------------------------------------------------------------------\n";

        printf("\n\n\n");
    }
}

int main()
{
    run_tests<float>();
    run_tests<double>();
}